
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define BLOCK_DIM 1024 // number of threads in a block
#define TABLE_SIZE 26
//#define DEBUG

typedef struct
{
    int score;
    int offset;
    int k;
    int seq_len;
    char seq[2000];
} Seq_Info;





__global__ void work_cuda(char* d_seq2, int seq2_len,char* d_seq1,int seq1_len,int score_table[][26],int table_size,int* arr_max_result);

int computeOnGPU(char* seq1, char* seq2 ,int seq1_length,int seq2_length, int score_table[][26],int table_size,int* max_arr);
__device__ void scan_plus(int *array, int size);
__device__ void scan_maxl(int *array, int size);
__device__ char next_char(char c);


__global__ void work_cuda(char* d_seq2, int seq2_len,char* d_seq1,int seq1_len,int score_table[][26],int table_size,int* arr_max_result)
{

    int tid = threadIdx.x;
    __shared__ int scores[2* BLOCK_DIM];

        if(threadIdx.x == 0 ){
            scores[0] = 0;
            for(int i = 0 ; i<seq2_len ; i++){
                scores[0] += score_table[d_seq1[i+blockIdx.x]-'A'][next_char(d_seq2[i])-'A'];
                 }
            }
      

            while(tid < 2*BLOCK_DIM - 1){
                if(tid< seq2_len){
                int Score = score_table[d_seq1[tid+blockIdx.x]-'A'][d_seq2[tid]-'A'];
                int ScoreM = score_table[d_seq1[tid+blockIdx.x]-'A'][next_char(d_seq2[tid])-'A'];
                scores[tid+1] = Score-ScoreM;
                }else{
                     scores[tid+1] = 0;
                }
                tid += BLOCK_DIM;
                
            }


    __syncthreads();
    scan_plus(scores, BLOCK_DIM*2);
    __syncthreads();
    scan_maxl(scores, BLOCK_DIM*2);

    __syncthreads();

    tid = threadIdx.x ;

    while(tid < 2*BLOCK_DIM ){
        if (tid == 0 && scores[0] == scores[2*BLOCK_DIM - 1]){
            arr_max_result[2*blockIdx.x] = scores[tid]; //score
            arr_max_result[2*blockIdx.x+1] = tid;//k
        }else if(tid != 0 && scores[tid-1] != scores[tid] && scores[tid] == scores[2*BLOCK_DIM - 1]){
            arr_max_result[2*blockIdx.x] = scores[tid]; //score
            arr_max_result[2*blockIdx.x+1] = tid;//k
        }
    tid += BLOCK_DIM;
    }


}



__device__ void scan_maxl(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v,v1;
        int tid =threadIdx.x + blockDim.x ;

        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        if (tid >= stride) {
            v1 = array[ tid- stride];
        }
        __syncthreads(); // wait untill all threads get to this line

        if (threadIdx.x >= stride && array[threadIdx.x] < v )
            array[threadIdx.x] = v;

        if (tid >= stride && array[tid] < v1 )
            array[tid] = v1;

        __syncthreads(); // wait untill all threads get to this line
     }
     
}


__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v,v1;
        int tid =threadIdx.x + blockDim.x;

        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }

        if (tid >= stride) {
            v1 = array[ tid- stride];
        }
        
        __syncthreads(); // wait untill all threads get to this line

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        if (tid >= stride)
            array[tid] += v1;   

        __syncthreads(); // wait untill all threads get to this line
     }
     
} 

int computeOnGPU(char* seq1, char* seq2 ,int seq1_length,int seq2_length, int score_table[][26],int table_size,int* max_arr)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
   
    // Allocate memory on GPU to copy the data from the host
  char* d_seq1;
    err = hipMalloc((void **)&d_seq1, seq1_length*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_seq1, seq1, seq1_length*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return(1);
    }
    
    char *d_seq2;
    err = hipMalloc((void **)&d_seq2, (seq2_length)*sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(-1);
    }
      err = hipMemcpy(d_seq2, seq2, (seq2_length)*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "(4)Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(-1);
    }
    
    // Allocate memory on GPU to copy the TABLE from the host
    int *d_table;
    err = hipMalloc((void **)&d_table, table_size*table_size*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 1;
    }

    // Copy TABLE from host to the GPU memory
    err = hipMemcpy(d_table, score_table, table_size*table_size*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return(1);
    }

    dim3 blockDim;
    blockDim.x = BLOCK_DIM;
    int blocks = seq1_length - seq2_length + 1;

    int* arr_max_result;
          err = hipMalloc((void **)&arr_max_result, 2*blocks*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(-1);
    }
   



    work_cuda<<<blocks, blockDim.x>>>(d_seq2,seq2_length,d_seq1, seq1_length,(int(*)[TABLE_SIZE])d_table,table_size,arr_max_result);
    /* note: next lines may be executed before the kernel is done */
    err = hipGetLastError();
    if (err != hipSuccess) {
         fprintf(stderr, "Failed to launch incrementByOne kernel -  %s\n", hipGetErrorString(err));
        return(1);
        }


  
    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(max_arr, arr_max_result, 2*blocks*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        return(1);
    }

#ifdef DEBUG
    
    for(int i = 0 ; i < blocks; i++){
        printf("offset  = %d, k = %d  max score:%d\n",i,max_arr[(i*2)+1],max_arr[i*2] );
    }

#endif


    // Free allocated memory on GPU
    if (hipFree(arr_max_result) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return(1);
    }
    if (hipFree(d_table) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return(1);
    }

    return 0;



}

__device__ char next_char(char c)
{
    c = (c == 'Z') ? 'A' : c+1;
    return c;
}